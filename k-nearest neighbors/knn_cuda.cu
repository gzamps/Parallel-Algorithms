/* knn cuda */


#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include <time.h>
#include <sys/time.h>



/*gia compiler 
export PATH=/Developer/NVIDIA/CUDA-5.5/bin:$PATH
 kai
export DYLD_LIBRARY_PATH=/Developer/NVIDIA/CUDA-5.5/lib:$DYLD_LIBRARY_PATH
*/



typedef struct{
  float *dataset;
  int leading_dim;
  int secondary_dim;
} knn_struct;


//#define BlockSize 512
//#define NUMBER_OF_SUBMATRICES 128

void error_message_fewer(){
	char *help = "Entered less than four arguments";

	char *help2 = "Error using knns: Four arguments required\n"
  "First: number of elements\n"
  "Second: number of attributes (dimensions)\n"
  "Third: numder of queries\n"
  "Fourth: number of nearest neighbours\n";

  printf("\e[1;34m %s \e[0m", help);
  	printf("\n");
  printf("\e[1;34m %s \e[0m", help2);
}

void error_message_more(){
	char *help = "Entered more than four arguments";

	char *help2 = "Error using knns: Four arguments required\n"
  "First: number of elements\n"
  "Second: number of attributes (dimensions)\n"
  "Third: numder of queries\n"
  "Fourth: number of nearest neighbours\n";

  printf("\e[1;34m %s \e[0m", help);
  	printf("\n");
  printf("\e[1;34m %s \e[0m", help2);
}

char* choose_data_file(int n){
	
	if (n==524288){
		return "base524288.bin";
	}else if(n==786432){
		return "base786432.bin";
	} else{
		return"base1048576.bin";
	}
	
	
}

char* choose_query_file(int q){
	
	
	if (q==1){
		return "query1.bin";
	}else if(q==100){
		return "query100.bin";
	}else if(q==200){
		return "query200.bin";
	}else if(q==300){
		return "query300.bin";
	}else if(q==400){
		return "query400.bin";
	}else if(q==500){
		return "query500.bin";
	}else if(q==600){
		return "query600.bin";
	}else if(q==700){
		return "query700.bin";
	}else if(q==800){
		return "query800.bin";
	}else if(q==900){
		return "query900.bin";
	}else {
		return "query1000.bin";
	}		
		
		
	
}

void save_distances(float* tmp_dataset, char *filename1,int n,int k){

  
  FILE *outfile;
  //int n = data2save->leading_dim;
  //int m = data2save->secondary_dim;
  //double *tmp_dataset = data2save->dataset;
  //unsigned int *tmp_members = data2save->members;

  printf("Saving data to files: "); printf(filename1);  printf("\n");

  /*===========Save to file 1===========*/
  if((outfile=fopen(filename1, "wb")) == NULL){
    printf("Can't open output file\n");
  }

  fwrite(tmp_dataset, sizeof(float), n*k, outfile);

  fclose(outfile);

}

void save_indexes(int* tmp_dataset, char *filename1,int n,int k){

  
  FILE *outfile;
  //int n = data2save->leading_dim;
  //int m = data2save->secondary_dim;
  //double *tmp_dataset = data2save->dataset;
  //unsigned int *tmp_members = data2save->members;

  printf("Saving data to files: "); printf(filename1); printf("\n");

  /*===========Save to file 1===========*/
  if((outfile=fopen(filename1, "wb")) == NULL){
    printf("Can't open output file\n");
  }

  fwrite(tmp_dataset, sizeof(int), n*k, outfile);

  fclose(outfile);

}


void cleanDevice(knn_struct *data){

  hipFree(data->dataset);

}
/*
__global__ void selection_2(float* distances,float* NNdist,int* NNidx,int numObjects,int numQueries,int k){
	
	extern __shared__ float data[];
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int current_elem = tid;
	int current_block = blockIdx.x;
	
	__syncthreads();
	data[threadIdx.x]= distances[tid] 
	
}
*/

__global__ void select_kernel_last(float* next_distances_2, int*  next_indexes_2,float* NNdist,int* NNidx,int k, int q,int bs,float* last_distances,int* last_indexes){

	extern __shared__ float shared_data[];
	
	float* sdata = (float*)shared_data;
	int* sindexes = (int*)&sdata[512];
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//int current_elem = tid;
	//int current_block = blockIdx.x;
	float temp_dist;
	int temp_idx;
	if (tid<bs){
	
	float kdist[8];
	int kidx[8];
	__syncthreads();
	//load elements in shared memory
	sdata[threadIdx.x] = next_distances_2[ tid ];
	sindexes[threadIdx.x] = next_indexes_2[ tid];
	
	__syncthreads();

	for (int neighbour = 0 ; neighbour < k ; neighbour ++){

	
		
		for (unsigned int s=1; s < blockDim.x; s *= 2) {
			
			int index = 2 * s * threadIdx.x;
			if (index < blockDim.x) {
				
				if (sdata[index]>sdata[index + s]){

					temp_dist=sdata[index];
					sdata[index]=sdata[index + s];
					sdata[index + s]=temp_dist;
			
					temp_idx=sindexes[index];
					sindexes[index]=sindexes[index + s];
					sindexes[index + s]=temp_idx;
				}
	
			}

__syncthreads();

		}

		if (threadIdx.x==0){
		kdist[neighbour]=sdata[0];
		kidx[neighbour]=sindexes[0];
		sdata[0]=FLT_MAX;
		}
	__syncthreads();	
	}


	if(threadIdx.x==0){
		for (int neighbour = 0 ; neighbour < k ; neighbour ++){
			NNdist[  q*k + neighbour ] = kdist[neighbour];
			last_distances[ neighbour ]=kdist[neighbour];
		//distances[q*numObjects + neighbour]=sdata[neighbour];
			NNidx[  q*k + neighbour ] = kidx[neighbour];
			last_indexes[neighbour ]=kidx[neighbour];
		}
	}
	}

}
__global__ void select_kernel_2(float* next_distances,int* next_indexes,float* next_distances_2,int* next_indexes_2,int numQueries,int k ){
	
	extern __shared__ float shared_data[];
	
	float* sdata = (float*)shared_data;
	int* sindexes = (int*)&sdata[512];
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//int current_elem = tid;
	//int current_block = blockIdx.x;
	float temp_dist;
	int temp_idx;
	
	
	float kdist[8];
	int kidx[8];

	__syncthreads();
	//load elements in shared memory
	sdata[threadIdx.x] = next_distances[ tid ];
	sindexes[threadIdx.x] = next_indexes[ tid ];
	
	__syncthreads();
	
	for (int neighbour = 0 ; neighbour < k ; neighbour ++){
		
		for (unsigned int s=1; s < blockDim.x; s *= 2) {
			
			int index = 2 * s * threadIdx.x;
			if (index < blockDim.x) {
				
				if (sdata[index]>sdata[index + s]){

					temp_dist=sdata[index];
					sdata[index]=sdata[index + s];
					sdata[index + s]=temp_dist;
			
					temp_idx=sindexes[index];
					sindexes[index]=sindexes[index + s];
					sindexes[index + s]=temp_idx;
				}
	
			}

		__syncthreads();

		}
		if (threadIdx.x==0){
		kdist[neighbour]=sdata[0];
		kidx[neighbour]=sindexes[0];
		sdata[0]=FLT_MAX;
		}
		__syncthreads();
	}


	if(threadIdx.x==0){
		for (int neighbour = 0 ; neighbour < k ; neighbour ++){
			next_distances_2[  k*blockIdx.x + neighbour ] = kdist[neighbour];
		//distances[q*numObjects + neighbour]=sdata[neighbour];
			next_indexes_2[ k*blockIdx.x + neighbour ] = kidx[neighbour];
		}
	}
	
	
}




__global__ void select_kernel(float* distances,float* next_distances,int* next_indexes,int numObjects,int k){
	
	extern __shared__ float shared_data[];
	
	float* sdata = (float*)shared_data; 
	 int* sindexes = (int*)&sdata[512]; 
	  
	
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//int current_elem = tid;
	//int current_block = blockIdx.x;
	float temp_dist;
	int temp_idx;
	
	
	float kdist[8];
	int kidx[8];

	
	
	__syncthreads();
	
	//load elements in shared memory
	sdata[threadIdx.x] = distances[ tid ];
	sindexes[threadIdx.x] = tid;
	//sdata[0]=FLT_MAX;
	
	
	__syncthreads();
	
	
	for (int neighbour = 0 ; neighbour < k ; neighbour ++){
		



		for (unsigned int s=1; s < blockDim.x; s *= 2) {
			
			int index = 2 * s * threadIdx.x;
			if (index < blockDim.x) {
				
				if (sdata[index]>sdata[index + s]){

					temp_dist=sdata[index];
					sdata[index]=sdata[index + s];
					sdata[index + s]=temp_dist;
			
					temp_idx=sindexes[index];
					sindexes[index]=sindexes[index + s];
					sindexes[index + s]=temp_idx;
				}
	
			}



		}
__syncthreads();
		if (threadIdx.x==0){
		kdist[neighbour]=sdata[0];
		kidx[neighbour]=sindexes[0];
		sdata[0]=FLT_MAX;
		


		}
		
__syncthreads();




	}


	if(threadIdx.x==0){
		for (int neighbour = 0 ; neighbour < k ; neighbour ++){
			next_distances[ k*blockIdx.x + neighbour ] = kdist[neighbour];
		//distances[q*numObjects + neighbour]=sdata[neighbour];
			next_indexes[ k*blockIdx.x + neighbour ] = kidx[neighbour];
		}
	}
	
	
}


__device__ float euclidean_distance_gpu(float *v1, float *v2, int attributes, int numObjects){

  float dist = 0;
  
#pragma unroll 2
  for( int i = 0; i < attributes; i++ ){
    float tmp = v2[i*numObjects] - v1[i];
    dist += tmp * tmp;
  }
  return dist;
}

__device__ float my_euclidean_distance_gpu(float *v1, float* v2, int attributes,int numObjects){

	float dist = 0;
//#pragma unroll 2
	for(int i=0; i<attributes; i++){
		float tmp= v2[i]-v1[i];
		dist += tmp * tmp;
	}
	return dist;
}


__global__ void calculate_distances_seperately(float* dataset ,float* queries ,float* distances,int numObjects,int numAttributes,int numQueries,int k,int q){
	
	extern __shared__ float querymeans[];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int current_elem = tid;
	float tmp_dist;
	
	if (tid<numObjects){
		
			tmp_dist=0;
			
			__syncthreads();
			if ( threadIdx.x < numAttributes){
				querymeans[threadIdx.x]= queries[q*numAttributes + threadIdx.x];
				}
			__syncthreads();
			//tmp_dist = euclidean_distance_gpu( querymeans, dataset + current_elem, numAttributes, numObjects);
			tmp_dist = my_euclidean_distance_gpu( querymeans, dataset + current_elem*numAttributes , numAttributes, numObjects);
			distances[tid]=tmp_dist;
		
		
	}
	

	
	
}
__global__ void clear_distances(float* distances, int numObjects){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid<numObjects){
		distances[tid]=0.0;
	}
}

void knns(knn_struct* d_training_set, knn_struct* d_query_set, float* d_NNdist, int* d_NNidx, int tk,float* distances){
	
	float* dataset=d_training_set->dataset;
	float* queries=d_query_set->dataset;
	int k=tk;
	float* NNdist=d_NNdist;
	int* NNidx=d_NNidx;
	int numObjects=d_training_set->secondary_dim;
	int numAttributes=d_training_set->leading_dim;
	int numQueries=d_query_set->secondary_dim;
	//size_t memory_free, memory_total;
	//int i;
	int BlockSize=512;

	
#ifdef TIMEONLY
  float elapsedTime_kernel;
  cudaEvent_t start_kernel, stop_kernel;
  cudaEventCreate(&start_kernel);
  cudaEventCreate(&stop_kernel);
#endif
	
  
printf("tha diairesw %d dia  %d = %d",numObjects , BlockSize , numObjects/BlockSize);
	int tmp_grid_size = numObjects/BlockSize;
  int tmp_block_size = numObjects<BlockSize ? numObjects:BlockSize; 

  

  printf("tmp_grid_size = %d \n",tmp_grid_size );
  dim3 grid(tmp_grid_size,1);
  dim3 threads(tmp_block_size, 1);
  printf(" \n");
 
  
  
#ifdef TIMEONLY
  cudaEventRecord(start_kernel, 0);
#endif

  
  
int blocks_reducing=numObjects/BlockSize; 
int blocks_reducing_2=blocks_reducing/BlockSize; 
  float* next_distances;
  float* next_distances_2;
  int* next_indexes;
  int* next_indexes_2;
/*
  float* test_nd=(float *)malloc(k*blocks_reducing*sizeof(float));
  int* test_ni=(int *)malloc(k*blocks_reducing*sizeof(int));
  float* test_nd_2=(float *)malloc(k*k*blocks_reducing_2*sizeof(float));
  int* test_ni_2=(int *)malloc(k*k*blocks_reducing_2*sizeof(int));
  */
  
  hipMalloc((void**)&next_indexes, k*blocks_reducing*sizeof(int));
  hipMalloc((void**)&next_distances, k*blocks_reducing*sizeof(float));
/*
int* next_indexes_s;
float* next_distances_s;
  cudaMalloc((void**)&next_indexes_s, k*blocks_reducing*sizeof(int));
  cudaMalloc((void**)&next_distances_s, k*blocks_reducing*sizeof(float));
*/
  hipMalloc((void**)&next_indexes_2, k*k*blocks_reducing_2*sizeof(int));
  hipMalloc((void**)&next_distances_2, k*k*blocks_reducing_2*sizeof(float));


  float* last_distances;
  int* last_indexes;
  hipMalloc((void**)&last_distances, k*sizeof(float));
  hipMalloc((void**)&last_indexes, k*sizeof(int));
  //float* testld=(float *)malloc(k*sizeof(float));
 // int* testid=(int *)malloc(k*sizeof(int));
  printf("knn starts \n");




//test
//  float *te=(float *)malloc(1*sizeof(float));
//float *te1=(float *)malloc(1*sizeof(float));
//float *te2=(float *)malloc(1*sizeof(float));
//float *t=(float *)malloc((numObjects-1000000)*sizeof(float));

//====== for each query =====//
for (int q=0; q<numQueries; q++){
 
 
  clear_distances<<<grid ,threads >>>(distances, numObjects);
  
  calculate_distances_seperately<<<grid , threads , numAttributes*sizeof(float)>>>(dataset, queries, distances, numObjects, numAttributes, numQueries, k, q);
 
/*
 cudaMemcpy(t, distances + 1000000, (numObjects-1000000)*sizeof(float), cudaMemcpyDeviceToHost);
  for(i=0;i< numObjects-1000000 -48000 ;i++){
	printf("stoixio[%d] = %f ,,,,, ", i+1000000 , t[ i ]);
}
  /*
  //cudaMemcpy(testdist, distances, numObjects*sizeof(float), cudaMemcpyDeviceToHost);
  for (i=500000;i<500500;i++){
  	printf("| %d-> %f |.",i, testdist[i]);
  }
  */
  //	cudaMemcpy(te, distances + 1000021, 1*sizeof(float), cudaMemcpyDeviceToHost);
  	//printf("meta to cal dist, te:%f\n",te[0]);
  select_kernel<<<  grid, threads, 2*BlockSize*sizeof(float) >>>(distances, next_distances, next_indexes, numObjects, k);
	


//cudaMemcpy(te1, next_distances + k*blocks_reducing- 5, 1*sizeof(float), cudaMemcpyDeviceToHost);
//printf("meta to cal dist, te1:%f\n",te1[0]);

/*
cudaMemcpy(test_ni, next_indexes, k*blocks_reducing*sizeof(int), cudaMemcpyDeviceToHost);
cudaMemcpy(test_nd, next_distances, k*blocks_reducing*sizeof(float), cudaMemcpyDeviceToHost);


	 for(i=0;i<k*blocks_reducing;i++){
	 	printf("next_indexes[%d]= %d me dist =%f\n", i,test_ni[i],test_nd[i]);
	 }
*/
	 dim3 newgrid((int)k*blocks_reducing/BlockSize,1);
	 dim3 newthreads((int)BlockSize,1);
	 
	 select_kernel_2<<<  newgrid, newthreads, 2*BlockSize*sizeof(float) >>>( next_distances, next_indexes, next_distances_2, next_indexes_2, numQueries, k);
	
	//cudaMemcpy(te2, distances, 1*sizeof(float), cudaMemcpyDeviceToHost);

	 //cudaMemcpy(test_ni_2, next_indexes_2, k*k*blocks_reducing_2*sizeof(int), cudaMemcpyDeviceToHost);
     //cudaMemcpy(test_nd_2, next_distances_2, k*k*blocks_reducing_2*sizeof(float), cudaMemcpyDeviceToHost);
/*
	for(i=0;i<k*k*blocks_reducing_2;i++){
	 	printf("next_indexes_2[%d]= %d me dist =%f \n", i,test_ni_2[i],test_nd_2[i]);
	 }
*/

	 dim3 lastgrid(1,1);
	 dim3 lastthreads((int)BlockSize,1); //2 -> 64 

	 select_kernel_last<<<  lastgrid, lastthreads, 2*BlockSize*sizeof(float) >>>( next_distances_2, next_indexes_2, NNdist, NNidx, k ,q, k*k*blocks_reducing_2, last_distances,last_indexes);
	 //cudaMemcpy(testld, last_distances, k*sizeof(float), cudaMemcpyDeviceToHost);
	// cudaMemcpy(testid, last_indexes, k*sizeof(int), cudaMemcpyDeviceToHost);
	 /*
	 for (int i = 0; i < k; i++)
	 {
	 	printf("last_distances= %f apo index= %d \n",testld[i],testid[i] );
	 }
	 */
  }
  
 


  
  printf("Done with knns\n");
 

  
#ifdef TIMEONLY
  cudaEventRecord(stop_kernel, 0);  
  cudaEventSynchronize(stop_kernel);
#endif
  
  
  
#ifdef TIMEONLY
  cudaEventElapsedTime(&elapsedTime_kernel, start_kernel, stop_kernel);
  printf("Time elapsed for kernel execution: %f ms\n", elapsedTime_kernel);
#endif
	
	
  hipFree(distances);
  hipFree(next_distances);
  hipFree(next_indexes);

#ifdef TIMEONLY
  cudaEventDestroy(start_kernel);
  cudaEventDestroy(stop_kernel);
#endif

}	
		





int main(int argc, char **argv){


  struct timeval first, second, lapsed;
  struct timezone tzp;
  size_t memory_free, memory_total;
  
  
hipMemGetInfo(&memory_free, &memory_total);
  printf("Totel memory: %zd, free memory: %zd\n", memory_total, memory_free);

  
 

  if(argc<5){
    error_message_fewer();
    return 0;
  }
  if (argc>5){
  	error_message_more();
  }


  int numObjects = atoi(argv[1]);
  if (numObjects<524288 || numObjects>1048576){
    printf("invalind number of objects\n");
    return 1;
  }
  int numAttributes = atoi(argv[2]);
  if (numAttributes<128 || numAttributes>128){
    printf("invalid number of attributes\n");
    return 2;
  }
  int numQueries = atoi(argv[3]);
  if ( numQueries<1 && (numQueries>=100) && ( (numQueries % 100) !=0 ) && ( numQueries >1000 ) ) {
    printf("invalid number of queries\n");
    return 3;
  }
  int k = atoi(argv[4]);
  if (k<1 || k>8){
    printf("invalid number of nearest neighbours\n");
    return 4;
  }

  char *dataset_file = choose_data_file(numObjects);
  char *query_file = choose_query_file(numQueries);



  printf("name file dataset: %s \n", dataset_file);
  printf("name file queries: %s \n",query_file );
  
  printf("objects: %d\n", numObjects);
  printf("dimentions: %d\n", numAttributes);
  printf("queries: %d\n", numQueries);
  printf("k: %d\n", k);
  
  char *file1 = "NN_distances_cuda.bin";
  char *file2 = "NN_indexes_cuda.bin";

  knn_struct training_set;
  knn_struct query_set;
  float *NNdist;
  int *NNidx;

  training_set.leading_dim = numAttributes;
  training_set.secondary_dim = numObjects;
  query_set.leading_dim = numAttributes;
  query_set.secondary_dim = numQueries;

  /*======== Memory allocation ======*/
  training_set.dataset = (float*)malloc(numObjects*numAttributes*sizeof(float));
  query_set.dataset = (float*)malloc(numQueries*numAttributes*sizeof(float));
  NNdist = (float*)malloc(numQueries*k*sizeof(float));
  NNidx = (int*)malloc(numQueries*k*sizeof(int));



  int i,j;


//===file data====//

  FILE *fp;
  float* data = (float*)malloc(numObjects*numAttributes*sizeof(float));

  fp = fopen( dataset_file , "rb");
  if(fp==NULL){printf("Error opening the file\n");}

  int w = fread(data, sizeof(float), numObjects*numAttributes, fp);
  if(w!=numObjects*numAttributes){printf("Error reading the data\n");}
  
  
 
  for (i=0 ; i<numObjects ; i++){
  	for (j=0 ; j< numAttributes ; j++ ){
  		//z= data[ i*numAttributes + j];
  		training_set.dataset[ i*numAttributes + j] = data[ i*numAttributes + j];
  	}
  	
  }
  
  fclose(fp);
  free(data);



//===file query ====//
FILE *fpq;
  float* queries = (float*)malloc(numQueries*numAttributes*sizeof(float));

  fpq = fopen(query_file, "rb");
  if(fpq==NULL){printf("Error opening the file\n");}

  int w2 = fread(queries, sizeof(float), numQueries*numAttributes, fpq);
  if(w2!=numQueries*numAttributes){printf("Error reading the data\n");}

  for (i=0 ; i<numQueries ; i++){
  		  for (j=0 ; j<numAttributes ; j++){
		query_set.dataset[ i*numAttributes + j] = queries[ i*numAttributes + j];
	  	
	  }
  }

  fclose(fpq);
  free(queries);
  /*===== Cuda Events===*/

  float elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // ==== device stuff ==== //
  hipMemGetInfo(&memory_free, &memory_total);
  printf("Total memory: %zd, free memory: %zd\n", memory_total, memory_free);
  
  knn_struct d_training_set;
  knn_struct d_query_set;
  float *d_NNdist;
  int *d_NNidx;
  float *d_distances;

  d_training_set.leading_dim = numAttributes;
  d_training_set.secondary_dim = numObjects;
  d_query_set.leading_dim = numAttributes;
  d_query_set.secondary_dim = numQueries;
  
 
  /*========= device memory allocation======*/

  hipMalloc((void**)&d_training_set.dataset, d_training_set.leading_dim*d_training_set.secondary_dim*sizeof(float));

  hipMalloc((void**)&d_query_set.dataset, d_query_set.leading_dim*d_query_set.secondary_dim*sizeof(float));

  hipMalloc((void**)&d_NNdist , d_query_set.secondary_dim*k*sizeof(float));
  
  hipMalloc((void**)&d_NNidx , d_query_set.secondary_dim*k*sizeof(int));
  
  
  hipMalloc((void**)&d_distances, d_training_set.secondary_dim*sizeof(float));
  
 

  hipMemcpy(d_training_set.dataset, training_set.dataset, training_set.leading_dim*training_set.secondary_dim*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_query_set.dataset, query_set.dataset, query_set.leading_dim*query_set.secondary_dim*sizeof(float), hipMemcpyHostToDevice);
  


 hipEventRecord(start, 0);
  

  knns(&d_training_set, &d_query_set, d_NNdist, d_NNidx, k, d_distances);
  
  
  hipMemcpy(NNdist, d_NNdist, k*query_set.secondary_dim*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(NNidx, d_NNidx, k*query_set.secondary_dim*sizeof(int), hipMemcpyDeviceToHost);
  
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);


  hipEventElapsedTime(&elapsedTime, start, stop);

#ifdef TIMEONLY
  printf("Time elapsed: %f ms\n", elapsedTime);
#endif

   
  printf("Time elapsed: %f ms\n", elapsedTime);
  
  
  gettimeofday(&second, &tzp);


  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("Time elapsed: %d.%06dsec\n", lapsed.tv_sec, lapsed.tv_usec); 

/*
 for (i=0;i<numQueries;i++){
	for (j=0; j<k; j++){
		printf("apostash %d query apo ton geirona %d = %f\n",i,NNidx[i*k + j],NNdist[i*k + j] );
	}
}
*/
  /*========save data============*/
  save_distances(NNdist, file1, numQueries, k);
  save_indexes(NNidx, file2, numQueries, k);
  

  /*==== clean device===*/
  cleanDevice(&d_training_set);
  cleanDevice(&d_query_set);


  hipEventDestroy(start);
  hipEventDestroy(stop);


  hipDeviceReset();
  
  return 1;
}
  
  
  